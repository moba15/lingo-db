#include "lingodb/runtime/GPU/Properties.h"
#include "lingodb/runtime/GPU/CUDA/CudaUtils.cuh"


std::string lingodb::runtime::gpu::getChipStr(uint32_t deviceId) {
   int major, minor;
   CHECK_CUDA_ERROR(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, deviceId));
   CHECK_CUDA_ERROR(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, deviceId));
   return "sm_" + std::to_string(major) + std::to_string(minor);
}